
#include <hip/hip_runtime.h>
#include <iostream>

// Simple CUDA kernel
__global__ void hello_kernel() {
    printf("Hello World from GPU thread %d\n", threadIdx.x);
}

int main() {
    std::cout << "Hello from CPU!" << std::endl;

    // Launch kernel with 5 threads
    hello_kernel<<<1, 5>>>();
    hipDeviceSynchronize(); // wait for GPU to finish

    return 0;
}

